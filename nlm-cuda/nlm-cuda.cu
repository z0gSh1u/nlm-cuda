#include <iostream>
#include <hip/hip_runtime.h>

__global__ void addTwoNumber(int a, int b, int *c) { *c = a + b; }

int main() {
  int *c;
  int h_c;

  hipMalloc(&c, sizeof(int));
  addTwoNumber<<<1, 1>>>(2, 3, c);

  hipMemcpy(&h_c, c, sizeof(int), hipMemcpyDeviceToHost);
  std::cout << h_c << " hello" << std::endl;
  return 0;
}